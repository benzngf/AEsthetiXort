#include "hip/hip_runtime.h"
#include "PixelSort.h"
#include <hip/hip_runtime.h>
#ifdef DEBUG
	#include <stdio.h>
	#include <math.h>
#endif
#include <thrust/sort.h>
#include <thrust/execution_policy.h>

__device__ __host__ int CeilDiv(int a, int b) { return (a-1)/b + 1; }
__device__ __host__ int CeilAlign(int a, int b) { return CeilDiv(a, b) * b; }
__device__ __host__ int MaxRGB(const int R, const int G, const int B) {
    if(R > G && R > B) return R;
    else if (G > R && G > B) return G;
    else if (B > R && B > G) return B;
    else return R; // R == G && R == B
}
__device__ __host__ int MinRGB(const int R, const int G, const int B) {
    if(R < G && R < B) return R;
    else if (G < R && G < B) return G;
    else if (B < R && B < G) return B;
    else return R; // R == G && R == B
}
__device__ __host__ int absolute(int x){
    if (x >= 0)
        return x;
    return -x;
}
__device__ __host__ int getLuminance(const int R, const int G, const int B) {
    int M = MaxRGB(R, G, B);
    int m = MinRGB(R, G, B);
    return (M + m) / 2;
}  
__device__ __host__ int getHue(const int R, const int G, const int B) {
    int M = MaxRGB(R, G, B);
    int m = MinRGB(R, G, B);
    int C = M - m;

    if (C == 0)
        return 0;
    else if (M == R)
        return 60 * ( ( (G - B) / C ) % 6 );
    else if (M == G)
        return 2 + ( (B - R) / C );
    else if (M == B)
        return 4 + ( (R - G) / C );
    return 0;
}
__device__ __host__ int getSaturation(const int R, const int G, const int B) {
    int M = MaxRGB(R, G, B);
    int m = MinRGB(R, G, B);
    int C = M - m;
    int L = (M + m) / 2;

    if (L == 1)
        return 0;
    else
        return C / (1 - absolute(2 * L - 1));
}


#ifdef DEBUG
#define debug_print(...) fprintf(stderr, __VA_ARGS__) 
#else
#define debug_print(...)
#endif

#define OUPUT_POINT_MAX 100

//#define PREDEBUG

// TODO: I think these code is GPU-unfriendly
// This shoulb be __device__
#ifdef PREDEBUG
__global__  void GetListToSort(
#else
__device__  void GetListToSort(
#endif
        PixelSortPatternParmLinear *linear, 
        const float x, const float y, 
        const float w, const float h, 
        int *order, int *point_cnt, float *output) {
    float delta[2], last[2];
    int cnt = 1;
    
    delta[0] = cos(linear->angle);
    delta[1] = sin(linear->angle);
    
    output[0] = x;
    output[1] = y;
    
    // prev
    last[0] = x - delta[0];
    last[1] = y - delta[1];
    while (cnt < OUPUT_POINT_MAX && last[0] > 0 && last[0] < w && last[1] > 0 && last[1] < h) {
        output[cnt*2] = last[0];
        output[cnt*2+1] = last[1];
        ++cnt;
        last[0] -= delta[0];
        last[1] -= delta[1];
    }

    *order = cnt-1;

    // next
    last[0] = x + delta[0];
    last[1] = y + delta[1];
    while (cnt < OUPUT_POINT_MAX && last[0] > 0 && last[0] < w && last[1] > 0 && last[1] < h) {
        output[cnt*2] = last[0];
        output[cnt*2+1] = last[1];
        ++cnt;
        last[0] += delta[0];
        last[1] += delta[1];
    }

    *point_cnt = cnt;
}


#ifndef PREDEBUG
__global__ void SortFromList(PixelSortPatternParmLinear *linear, 
    const Pixel *input, Pixel *output, 
    const int w, const int h,
    const PixelSortBy sort_by) {

    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;

    const int pixelid = y * w + x;

    if (x < w && y < h) {
        // Get a list for sorting
        const float pixelx = x + 0.5;
        const float pixely = y + 0.5;

        int point_cnt_gpu;
        int order_gpu;
        float sort_list_gpu[2*OUPUT_POINT_MAX];


        GetListToSort(linear, pixelx, pixely, (float)w, (float)h, &order_gpu, &point_cnt_gpu, sort_list_gpu);

        // Sorting: preprocessing
        int converted_sort_list_gpu[OUPUT_POINT_MAX];
        Pixel pixel_list_gpu[OUPUT_POINT_MAX];
        // Fill sorting pixel
        for (int i = 0; i < point_cnt_gpu; i++){
            int ind = (int)floorf(sort_list_gpu[2*i+1])*w + (int)floorf(sort_list_gpu[2*i]);
            pixel_list_gpu[i].r = input[ind].r;
            pixel_list_gpu[i].g = input[ind].g;
            pixel_list_gpu[i].b = input[ind].b;
            pixel_list_gpu[i].a = input[ind].a;

        }
        // Fill sorting key
        switch (sort_by){

            case PSB_R:
                for (int i = 0; i < point_cnt_gpu; i++) {
                    int ind = (int)floorf(sort_list_gpu[2*i+1])*w + (int)floorf(sort_list_gpu[2*i]);
                    converted_sort_list_gpu[i] = input[ind].r;
                }
                break;
            case PSB_G:
                for (int i = 0; i < point_cnt_gpu; i++) {
                    int ind = (int)floorf(sort_list_gpu[2*i+1])*w + (int)floorf(sort_list_gpu[2*i]);
                    converted_sort_list_gpu[i] = input[ind].g;
                }
                break;
            case PSB_B:
                for (int i = 0; i < point_cnt_gpu; i++) {
                    int ind = (int)floorf(sort_list_gpu[2*i+1])*w + (int)floorf(sort_list_gpu[2*i]);
                    converted_sort_list_gpu[i] = input[ind].b;
                }
                break;
            case PSB_Hue:
                for (int i = 0; i < point_cnt_gpu; i++) {
                    int ind = (int)floorf(sort_list_gpu[2*i+1])*w + (int)floorf(sort_list_gpu[2*i]);
                    converted_sort_list_gpu[i] = getHue(input[ind].r, input[ind].g, input[ind].b);
                }
                break;
            case PSB_Saturation:
                for (int i = 0; i < point_cnt_gpu; i++) {
                    int ind = (int)floorf(sort_list_gpu[2*i+1])*w + (int)floorf(sort_list_gpu[2*i]);
                    converted_sort_list_gpu[i] = getSaturation(input[ind].r, input[ind].g, input[ind].b);
                }
                break;
            case PSB_Luminance:
                for (int i = 0; i < point_cnt_gpu; i++) {
                    int ind = (int)floorf(sort_list_gpu[2*i+1])*w + (int)floorf(sort_list_gpu[2*i]);
                    converted_sort_list_gpu[i] = getLuminance(input[ind].r, input[ind].g, input[ind].b);
                }
                break;
            default:
                break;
        }

        // Sort
        //thrust::sort_by_key(thrust::device, converted_sort_list_gpu, converted_sort_list_gpu + point_cnt_gpu, pixel_list_gpu);
        
        for (int i = 0; i < point_cnt_gpu; i++) {
            for (int j = 0; j < point_cnt_gpu - i; j++){
                if (converted_sort_list_gpu[j] > converted_sort_list_gpu[j+1]) {
                    int temp = converted_sort_list_gpu[j];
                    converted_sort_list_gpu[j] = converted_sort_list_gpu[j+1];
                    converted_sort_list_gpu[j+1] = temp;

                    temp = pixel_list_gpu[j].r;
                    pixel_list_gpu[j].r = pixel_list_gpu[j+1].r;
                    pixel_list_gpu[j+1].r = temp;

                    temp = pixel_list_gpu[j].g;
                    pixel_list_gpu[j].g = pixel_list_gpu[j+1].g;
                    pixel_list_gpu[j+1].g = temp;

                    temp = pixel_list_gpu[j].b;
                    pixel_list_gpu[j].b = pixel_list_gpu[j+1].b;
                    pixel_list_gpu[j+1].b = temp;

                    temp = pixel_list_gpu[j].a;
                    pixel_list_gpu[j].a = pixel_list_gpu[j+1].a;
                    pixel_list_gpu[j+1].a = temp;
                }
            }
        }
        // Fill value: order

        output[pixelid].r = pixel_list_gpu[order_gpu].r;
        output[pixelid].g = pixel_list_gpu[order_gpu].g;
        output[pixelid].b = pixel_list_gpu[order_gpu].b;
        output[pixelid].a = pixel_list_gpu[order_gpu].a;



    }



}
#endif
/*
__global__ sort() {
    const int x = ;
    const int y = ;
    const int pixelid = y*w + x;
    
    if (x and y are in the valid location) {
        const float pixelx = x + 0.5, pixely = y + 0.5;

        int order, point_cnt;
        float output[OUPUT_POINT_MAX*2];

        GetListToSort( PixelSortPatternParmLinear *linear, pixelx, pixely, w,  h, &order, &point_cnt, &output);

           //point -> SortBy


        thrust::sort(output);

        background[pixelid] = output[order];

    }
}
*/
/*(input image, image width, image height, output image(to fill),
sort by? (RGB...), threshold_min, threshold, max, reverse?
pattern parameter, do antialiasing?, sort alpha?)*/

void PixelSortGPU(const Pixel *input, int width, int height, Pixel *output,
	PixelSortBy sort_by, float threshold_min, float threshold_max, bool reverse_sort_order,
	PixelSortPatternParm *pattern_parm, bool anti_aliasing, bool sort_alpha) {

    PixelSortPatternParm *pattern_parm_gpu = nullptr;
	switch (pattern_parm->pattern) {
        case PSP_Linear:
            {
            debug_print("PSP_Linear (%d)\n", pattern_parm->pattern);
            debug_print("angle: %f\n", ((PixelSortPatternParmLinear *)pattern_parm)->angle);
            hipMalloc(&pattern_parm_gpu, sizeof(PixelSortPatternParmLinear));
            hipMemcpy(pattern_parm_gpu, pattern_parm, sizeof(PixelSortPatternParmLinear), hipMemcpyHostToDevice);
#ifndef PREDEBUG            
            dim3 gdim(CeilDiv(width, 32), CeilDiv(height, 16)), bdim(32, 16);
            SortFromList<<<gdim, bdim>>>((PixelSortPatternParmLinear *)pattern_parm_gpu, 
                                            input, output, width, height, sort_by);
#endif            
            break;
            }
        case PSP_Radial_Spin:
            debug_print("PSP_Radial_Spin (%d)\n", pattern_parm->pattern);
            debug_print("center: (%f, %f)\n", 
                    ((PixelSortPatternParmRadialSpin *)pattern_parm)->center[0],
                    ((PixelSortPatternParmRadialSpin *)pattern_parm)->center[1]);
            debug_print("WHRatio: %f\n", ((PixelSortPatternParmRadialSpin *)pattern_parm)->WHRatio);
            debug_print("rotation: %f\n", ((PixelSortPatternParmRadialSpin *)pattern_parm)->rotation);
            hipMalloc(&pattern_parm_gpu, sizeof(PixelSortPatternParmRadialSpin));
            hipMemcpy(&pattern_parm_gpu, pattern_parm, sizeof(PixelSortPatternParmRadialSpin), hipMemcpyHostToDevice);
            break;
        case PSP_Polygon:
            debug_print("PSP_Polygon (%d)\n", pattern_parm->pattern);
            debug_print("center: (%f, %f)\n", 
                    ((PixelSortPatternParmPolygon *)pattern_parm)->center[0],
                    ((PixelSortPatternParmPolygon *)pattern_parm)->center[1]);
            debug_print("numSides: %d\n", ((PixelSortPatternParmPolygon *)pattern_parm)->numSides);
            debug_print("WHRatio: %f\n", ((PixelSortPatternParmPolygon *)pattern_parm)->WHRatio);
            debug_print("rotation: %f\n", ((PixelSortPatternParmPolygon *)pattern_parm)->rotation);
            hipMalloc(&pattern_parm_gpu, sizeof(PixelSortPatternParmPolygon));
            hipMemcpy(&pattern_parm_gpu, pattern_parm, sizeof(PixelSortPatternParmPolygon), hipMemcpyHostToDevice);
            break;
        case PSP_Spiral:
            debug_print("PSP_Spiral (%d)\n", pattern_parm->pattern);
            debug_print("center: (%f, %f)\n", 
                    ((PixelSortPatternParmSpiral *)pattern_parm)->center[0],
                    ((PixelSortPatternParmSpiral *)pattern_parm)->center[1]);
            debug_print("curveAngle: %f\n", ((PixelSortPatternParmSpiral *)pattern_parm)->curveAngle);
            debug_print("WHRatio: %f\n", ((PixelSortPatternParmSpiral *)pattern_parm)->WHRatio);
            debug_print("rotation: %f\n", ((PixelSortPatternParmSpiral *)pattern_parm)->rotation);
            hipMalloc(&pattern_parm_gpu, sizeof(PixelSortPatternParmSpiral));
            hipMemcpy(&pattern_parm_gpu, pattern_parm, sizeof(PixelSortPatternParmSpiral), hipMemcpyHostToDevice);
            break;
        case PSP_Sine: 
        case PSP_Triangle: 
        case PSP_Saw_Tooth:
            debug_print("PSP_Sine (%d)\n", pattern_parm->pattern);
            debug_print("waveLength: %f\n", ((PixelSortPatternParmWave *)pattern_parm)->waveLength);
            debug_print("waveheight: %f\n", ((PixelSortPatternParmWave *)pattern_parm)->waveHeight);
            debug_print("rotation: %f\n", ((PixelSortPatternParmWave *)pattern_parm)->rotation);
            hipMalloc(&pattern_parm_gpu, sizeof(PixelSortPatternParmWave));
            hipMemcpy(&pattern_parm_gpu, pattern_parm, sizeof(PixelSortPatternParmWave), hipMemcpyHostToDevice);
            break;
        case PSP_Optical_Flow:
           /* debug_print("PSP_Optical_Flow (%d)\n", pattern_parm->pattern);
            hipMalloc(&pattern_parm_gpu, sizeof(PixelSortPatternParmOpFlow));
            hipMemcpy(&pattern_parm_gpu, pattern_parm, sizeof(PixelSortPatternParmOpFlow), hipMemcpyHostToDevice);*/
            break;

        default:
            break;
	}

    // Test code
#ifdef PREDEBUG    
    {
        int *point_cnt = (int *)malloc(sizeof(int));
        int *point_cnt_gpu;

        int *order = (int *)malloc(sizeof(int));
        int *order_gpu;

        float *sort_list = (float *)malloc(sizeof(float)*OUPUT_POINT_MAX*2);
        float *sort_list_gpu;

        hipMalloc(&sort_list_gpu, sizeof(float) * OUPUT_POINT_MAX*2);
        hipMalloc(&point_cnt_gpu, sizeof(int));
        hipMalloc(&order_gpu, sizeof(int));

        debug_print("w:%d, h:%d\n", width, height);
        GetListToSort<<<1, 1>>>((PixelSortPatternParmLinear *)pattern_parm_gpu, 500.0f, 300.0f, (float)width, (float)height, order_gpu, point_cnt_gpu, sort_list_gpu);

        hipMemcpy(point_cnt, point_cnt_gpu, sizeof(int), hipMemcpyDeviceToHost);
        hipMemcpy(order, order_gpu, sizeof(int), hipMemcpyDeviceToHost);
        hipMemcpy(sort_list, sort_list_gpu, sizeof(float)*2*(*point_cnt), hipMemcpyDeviceToHost);

        printf("%d, %d\n", *order, *point_cnt);
        for (int i = 0; i < *point_cnt; ++i) {
            printf("%d: (%f, %f)\n", i, sort_list[2*i], sort_list[2*i+1]);
        }

        free(sort_list);
        hipFree(sort_list_gpu);
        free(order);
        hipFree(order_gpu);
        free(point_cnt);
        hipFree(point_cnt_gpu);
    }
#endif
    

    //hipMemcpy(output, input, width*height*sizeof(Pixel), hipMemcpyDeviceToDevice);
    hipFree(pattern_parm_gpu);
}

///////////////////////CODE TO DETECT CUDA/////////////////////////////////
/*
for CUDA supporting...

After Effects does not support CUDA based code, if the following functions are there in CUDA programe.
Please remember dont use the below functions while writing the CUDA code.
1) printf()
2) puts()
3) getchar()
4) exit()
5) All timer related functions like cutCreateTimer()
6) dont use CUDA_SAFE_CALL and CUT_SAFE_CALL macros ( which are mostly used in CUDA SDK samples ).
7) CUT_DEVICE_INIT and CUT_EXIT macros
*/

extern "C"
int callCudaFunc();

__global__
void Kernal(int a, int b, int* sum)
{
	*sum = a + b;
}

extern "C"
int callCudaFunc()
{
	// initialise Device
	int deviceCount;
	hipGetDeviceCount(&deviceCount);

	if (deviceCount < 0)
		return 0;

	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, deviceCount);

	// Allocate memory on Device
	int* D_sum = NULL;
	hipError_t error = hipMalloc((void**)&D_sum, sizeof(int) * 1);


	// global function
	Kernal <<<1, 1 >> >(1, 2, D_sum);


	// copy data from Device memory to host memory
	int H_sum = 0;
	error = hipMemcpy(&H_sum, D_sum, sizeof(int) * 1, hipMemcpyDeviceToHost);

	// return 
	return H_sum;
}

void cuLayerMemMove(Pixel* GPUinputMem_CPU, Pixel* &GPUinputMem_GPUIn, Pixel* &GPUinputMem_GPUOut, int size,int dir) {
	if (dir == 0) {
		hipMalloc((void**)&GPUinputMem_GPUIn, sizeof(Pixel)*size);
		hipMalloc((void**)&GPUinputMem_GPUOut, sizeof(Pixel)*size);
		hipMemcpy(GPUinputMem_GPUIn, GPUinputMem_CPU, sizeof(Pixel)*size, hipMemcpyHostToDevice);
	}
	else if (dir == 1) {
		hipMemcpy(GPUinputMem_CPU, GPUinputMem_GPUOut, sizeof(Pixel)*size, hipMemcpyDeviceToHost);
		hipFree(GPUinputMem_GPUIn);
		hipFree(GPUinputMem_GPUOut);
	}
}
