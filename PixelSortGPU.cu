#include "hip/hip_runtime.h"
#include "PixelSort.h"
#include <hip/hip_runtime.h>
#ifdef DEBUG
	#include <stdio.h>
	#include <math.h>
#endif
#include <thrust/sort.h>
#include <thrust/execution_policy.h>

__device__ __host__ int CeilDiv(int a, int b) { return (a-1)/b + 1; }
__device__ __host__ int CeilAlign(int a, int b) { return CeilDiv(a, b) * b; }
__device__ __host__ __forceinline__ float MaxRGB(const float R, const float G, const float B) {
    if(R > G && R > B) return R;
    else if (G > R && G > B) return G;
    else if (B > R && B > G) return B;
    else return R; // R == G && R == B
}
__device__ __host__ __forceinline__ float MinRGB(const float R, const float G, const float B) {
    if(R < G && R < B) return R;
    else if (G < R && G < B) return G;
    else if (B < R && B < G) return B;
    else return R; // R == G && R == B
}
__device__ __host__ __forceinline__ float absolute(float x){
    if (x >= 0)
        return x;
    return -x;
}
__device__ __host__ __forceinline__ float angle_nomralize(float x){
    if (x < 0.0)
        return x + 6.0;
    return x;
}
__device__ __host__ float getLuminance(const float R, const float G, const float B) {
    return (MaxRGB(R, G, B) + MinRGB(R, G, B)) / 510.f;
}  
__device__ __host__ float getHue(const float R, const float G, const float B) {
    float M = MaxRGB(R, G, B);
    float m = MinRGB(R, G, B);
    float C = M - m;

    float Result = 0.0f;
    if (C > -0.1f && C < 0.1f) //C == 0.0f
        Result = 0.0f;
    else if (M == R)
        Result = angle_nomralize((G - B) / C);
    else if (M == G)
        Result = 2.0f + ( (B - R) / C );
    else if (M == B)
        Result = 4.0f + ( (R - G) / C );
    return 60.0f*Result;
}
__device__ __host__ float getSaturation(const float R, const float G, const float B) {
    float C = (MaxRGB(R, G, B) - MinRGB(R, G, B)) / 255.f;
    float L = getLuminance(R, G, B);

    if (C == 0.f) return 0.f;
    else if (L <= 0.5f) return (C > (2.f * L))? 1.0 : (C / (2.f * L));
    else return (C > (2.f - 2.f * L))? 1.0 : C / (2.f - 2.f * L);
}
// For the following 3 functions, adapted from: https://www.jiuzhang.com/solutions/kth-largest-element/
__device__ __host__ int kthLargestPartition(int l, int r, Pixel pixel_array[]) {

    int left = l, right = r;
    Pixel temp = pixel_array[left];
    float pivot = temp.key;
       

    while (left < right) {
        while (left < right && pixel_array[right].key >= pivot) {
            right--;
        }
        pixel_array[left] = pixel_array[right];
        while (left < right && pixel_array[left].key <= pivot) {
           left++;
        }
        pixel_array[right] = pixel_array[left];
    }        

    pixel_array[left] = temp;

    return left;         

}
__device__ __host__ int kthLargestInternal(int l, int r, int k, Pixel pixel_array[]) {
    if (l == r)
        return l;

    int position = kthLargestPartition(l, r, pixel_array);
    if (position + 1 == k)
        return position;
    else if (position + 1 < k)
        return kthLargestInternal(position + 1, r, k, pixel_array);
    else
        return kthLargestInternal(l, position - 1, k, pixel_array);
}
__device__ __host__ int kthLargest(int k, int length, Pixel pixel_array[]) {
    if (length == 0 || k <= 0)
        return -1;
    return kthLargestInternal(0, length - 1, length - k + 1, pixel_array);
}  


#ifdef DEBUG
#define debug_print(...) fprintf(stderr, __VA_ARGS__) 
#else
#define debug_print(...)
#endif

#define OUPUT_POINT_MAX 5000


// TODO: I think these code is GPU-unfriendly
__device__  void GetListToSort(
        const Pixel *input,
        PixelSortPatternParmLinear *linear, 
        const float x, const float y, 
        const float w, const float h, 
        int *order, int *point_cnt, Pixel *output) {
    float delta[2], last[2];
    int cnt = 1;
    
    delta[0] = cos(linear->angle);
    delta[1] = sin(linear->angle);
    
#ifdef SHOW_SORT
#define PIXELXY(x, y) (input[int(x) + int(y)*int(w)])
#define OUTPUTXY(x, y) (output[int(x) + int(y)*int(w)])
#define UPDATE_OUTPUT(x, y, red, green, blue) \
    OUTPUTXY(x, y).r = PIXELXY(x, y).r; \
    OUTPUTXY(x, y).g = PIXELXY(x, y).g; \
    OUTPUTXY(x, y).b = PIXELXY(x, y).b; 

    UPDATE_OUTPUT(x, y, 255, 255, 255);
    
    // prev
    last[0] = x - delta[0];
    last[1] = y - delta[1];
    while (cnt < OUPUT_POINT_MAX && 
           last[0] > 0 && last[0] < w && 
           last[1] > 0 && last[1] < h &&
           PIXELXY(last[0], last[1]).key >= 0.0f) {
        // TODO: AA here
        UPDATE_OUTPUT(last[0], last[1], 255, 0, 0);
        ++cnt;
        last[0] -= delta[0];
        last[1] -= delta[1];
    }

    *order = cnt-1;

    // next
    last[0] = x + delta[0];
    last[1] = y + delta[1];
    while (cnt < OUPUT_POINT_MAX && 
           last[0] > 0 && last[0] < w && 
           last[1] > 0 && last[1] < h &&
           PIXELXY(last[0], last[1]).key >= 0.0f) {
        // TODO: AA here
        UPDATE_OUTPUT(last[0], last[1], 0, 0, 255);
        ++cnt;
        last[0] += delta[0];
        last[1] += delta[1];
    }
#undef PIXELXY
#else
#define PIXELXY(x, y) (input[int(x) + int(y)*int(w)])
    output[0] = PIXELXY(x, y);
    
    // prev
    last[0] = x - delta[0];
    last[1] = y - delta[1];
    while (cnt < OUPUT_POINT_MAX && 
           last[0] > 0 && last[0] < w && 
           last[1] > 0 && last[1] < h &&
           PIXELXY(last[0], last[1]).key >= 0.0f) {
        // TODO: AA here
        output[cnt] = PIXELXY(last[0], last[1]);
        ++cnt;
        last[0] -= delta[0];
        last[1] -= delta[1];
    }

    *order = cnt-1;

    // next
    last[0] = x + delta[0];
    last[1] = y + delta[1];
    while (cnt < OUPUT_POINT_MAX && 
           last[0] > 0 && last[0] < w && 
           last[1] > 0 && last[1] < h &&
           PIXELXY(last[0], last[1]).key >= 0.0f) {
        // TODO: AA here
        output[cnt] = PIXELXY(last[0], last[1]);
        ++cnt;
        last[0] += delta[0];
        last[1] += delta[1];
    }
#undef PIXELXY
#endif

    *point_cnt = cnt;
}

/*
   Assume the domain of threshold_min, threshold_max is [0.0, 100.0]
   TODO: this is a naive version (use branches)
*/
__device__ float Map01(
        float x, 
        float min, float max, 
        float threshold_min, float threshold_max) {
    float len = max - min;
    threshold_min = min + (threshold_min/100.0f)*len;
    threshold_max = min + (threshold_max/100.0f)*len;

    if (threshold_max < threshold_min) {
        if (x > threshold_max && x < threshold_min) {
            return -1.0f;
        }
        float rhalf = max - threshold_min;
        float lhalf = threshold_max - min;
        float p = (x > threshold_max)? threshold_min : min;
        float offset = (x > threshold_max)? 0 : rhalf;

        return ((x - p) + offset) / (rhalf+lhalf);
    } else {
        if (x > threshold_max || x < threshold_min) {
            return -1.0f;
        }
        return (x - threshold_min) / (threshold_max - threshold_min);
    }
}

__global__ void ComputeKey(
        const PixelSortBy sort_by, 
        const int w, const int h,
        const float threshold_min, const float threshold_max,
#ifdef SHOW_SELECT
        Pixel *inout, Pixel *output)
#else
        Pixel *inout)
#endif
{
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < w && y < h) {
        const int pixelid = y * w + x;
        Pixel *cur = inout + pixelid;
        if (cur->a == 0.0f) {
            cur->key = -1.0f;
            return;
        }

        float min, max;
        switch (sort_by) {
            case PSB_R:
                cur->key = cur->r;
                min = 0;
                max = 255;
                break;
            case PSB_G:
                cur->key = cur->g;
                min = 0;
                max = 255;
                break;
            case PSB_B:
                cur->key = cur->b;
                min = 0;
                max = 255;
                break;
            case PSB_Hue:
                cur->key = getHue(cur->r, cur->g, cur->b);
                min = 0;
                max = 360;
                break;
            case PSB_Saturation:
                cur->key = getSaturation(cur->r, cur->g, cur->b);
                min = 0;
                max = 1;
                break;
            case PSB_Luminance:
                cur->key = getLuminance(cur->r, cur->g, cur->b);
                min = 0;
                max = 1;
                break;
            default:
                break;
        }
        cur->key = Map01(cur->key, min, max, threshold_min, threshold_max);
        /*
        if (cur->key > 0.0f) {
            cur->r = 255.0f;
            cur->g /= 2;
            cur->b /= 2;
        }
        */


#ifdef SHOW_SELECT
        if (cur->key >= 0.0f) {
            output[pixelid].r = output[pixelid].g = output[pixelid].b = 255.0f;
        }
#endif
    }
}

/*(input image, image width, image height, output image(to fill),
sort by? (RGB...), threshold_min, threshold, max, reverse?
pattern parameter, do antialiasing?, sort alpha?)*/

template <typename Parm>
__global__ void SortFromList(Parm *parm, 
    const Pixel *input, Pixel *output, 
    const int w, const int h) {

    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;

    const int pixelid = y * w + x;

#ifdef SHOW_SORT
    if (x == w/2 && y == h/2)
#else
    if (x < w && y < h)
#endif
    {
        const float pixelx = x + 0.5;
        const float pixely = y + 0.5;

        int point_cnt_gpu;
        int order_gpu;


        Pixel pixel_list_gpu[OUPUT_POINT_MAX];

#ifdef SHOW_SORT
        GetListToSort(input, parm, pixelx, pixely, (float)w, (float)h, &order_gpu, &point_cnt_gpu, output);
        return;
#else
        GetListToSort(input, parm, pixelx, pixely, (float)w, (float)h, &order_gpu, &point_cnt_gpu, pixel_list_gpu);
#endif
        // Sorting


#ifndef SORT_TEST

        int search_index = kthLargest(point_cnt_gpu - order_gpu, point_cnt_gpu, pixel_list_gpu);
#else
        for (int i = 0; i < point_cnt_gpu; i++) {
            for (int j = 0; j < point_cnt_gpu - i; j++){
                if (pixel_list_gpu[j].key > pixel_list_gpu[j+1].key) {
                    Pixel temp = pixel_list_gpu[j];
                    pixel_list_gpu[j] = pixel_list_gpu[j+1];
                    pixel_list_gpu[j+1] = temp;
                }
            }
        }
        int search_index = order_gpu;
#endif

        // Fill value: order
        output[pixelid].r = pixel_list_gpu[search_index].r;
        output[pixelid].g = pixel_list_gpu[search_index].g;
        output[pixelid].b = pixel_list_gpu[search_index].b;
        output[pixelid].a = pixel_list_gpu[search_index].a;
    }
}

void PixelSortGPU(Pixel *input, int width, int height, Pixel *output,
	PixelSortBy sort_by, float threshold_min, float threshold_max, bool reverse_sort_order,
	PixelSortPatternParm *pattern_parm, bool anti_aliasing, bool sort_alpha) {

    dim3 gdim(CeilDiv(width, 32), CeilDiv(height, 16)), bdim(32, 16);

#ifdef SHOW_SELECT
    ComputeKey<<<gdim, bdim>>>(sort_by, width, height, threshold_min, threshold_max, input, output);
    return;
#else
    ComputeKey<<<gdim, bdim>>>(sort_by, width, height, threshold_min, threshold_max, input);
#endif

    PixelSortPatternParm *pattern_parm_gpu = nullptr;
    switch (pattern_parm->pattern) {
        case PSP_Linear:
            {
            debug_print("PSP_Linear (%d)\n", pattern_parm->pattern);
            debug_print("angle: %f\n", ((PixelSortPatternParmLinear *)pattern_parm)->angle);
            hipMalloc(&pattern_parm_gpu, sizeof(PixelSortPatternParmLinear));
            hipMemcpy(pattern_parm_gpu, pattern_parm, sizeof(PixelSortPatternParmLinear), hipMemcpyHostToDevice);
            SortFromList<<<gdim, bdim>>>((PixelSortPatternParmLinear *)pattern_parm_gpu, input, output, width, height);
            break;
            }
        case PSP_Radial_Spin:
            debug_print("PSP_Radial_Spin (%d)\n", pattern_parm->pattern);
            debug_print("center: (%f, %f)\n", 
                    ((PixelSortPatternParmRadialSpin *)pattern_parm)->center[0],
                    ((PixelSortPatternParmRadialSpin *)pattern_parm)->center[1]);
            debug_print("WHRatio: %f\n", ((PixelSortPatternParmRadialSpin *)pattern_parm)->WHRatio);
            debug_print("rotation: %f\n", ((PixelSortPatternParmRadialSpin *)pattern_parm)->rotation);
            hipMalloc(&pattern_parm_gpu, sizeof(PixelSortPatternParmRadialSpin));
            hipMemcpy(&pattern_parm_gpu, pattern_parm, sizeof(PixelSortPatternParmRadialSpin), hipMemcpyHostToDevice);
            break;
        case PSP_Polygon:
            debug_print("PSP_Polygon (%d)\n", pattern_parm->pattern);
            debug_print("center: (%f, %f)\n", 
                    ((PixelSortPatternParmPolygon *)pattern_parm)->center[0],
                    ((PixelSortPatternParmPolygon *)pattern_parm)->center[1]);
            debug_print("numSides: %d\n", ((PixelSortPatternParmPolygon *)pattern_parm)->numSides);
            debug_print("WHRatio: %f\n", ((PixelSortPatternParmPolygon *)pattern_parm)->WHRatio);
            debug_print("rotation: %f\n", ((PixelSortPatternParmPolygon *)pattern_parm)->rotation);
            hipMalloc(&pattern_parm_gpu, sizeof(PixelSortPatternParmPolygon));
            hipMemcpy(&pattern_parm_gpu, pattern_parm, sizeof(PixelSortPatternParmPolygon), hipMemcpyHostToDevice);
            break;
        case PSP_Spiral:
            debug_print("PSP_Spiral (%d)\n", pattern_parm->pattern);
            debug_print("center: (%f, %f)\n", 
                    ((PixelSortPatternParmSpiral *)pattern_parm)->center[0],
                    ((PixelSortPatternParmSpiral *)pattern_parm)->center[1]);
            debug_print("curveAngle: %f\n", ((PixelSortPatternParmSpiral *)pattern_parm)->curveAngle);
            debug_print("WHRatio: %f\n", ((PixelSortPatternParmSpiral *)pattern_parm)->WHRatio);
            debug_print("rotation: %f\n", ((PixelSortPatternParmSpiral *)pattern_parm)->rotation);
            hipMalloc(&pattern_parm_gpu, sizeof(PixelSortPatternParmSpiral));
            hipMemcpy(&pattern_parm_gpu, pattern_parm, sizeof(PixelSortPatternParmSpiral), hipMemcpyHostToDevice);
            break;
        case PSP_Sine: 
        case PSP_Triangle: 
        case PSP_Saw_Tooth:
            debug_print("PSP_Sine (%d)\n", pattern_parm->pattern);
            debug_print("waveLength: %f\n", ((PixelSortPatternParmWave *)pattern_parm)->waveLength);
            debug_print("waveheight: %f\n", ((PixelSortPatternParmWave *)pattern_parm)->waveHeight);
            debug_print("rotation: %f\n", ((PixelSortPatternParmWave *)pattern_parm)->rotation);
            hipMalloc(&pattern_parm_gpu, sizeof(PixelSortPatternParmWave));
            hipMemcpy(&pattern_parm_gpu, pattern_parm, sizeof(PixelSortPatternParmWave), hipMemcpyHostToDevice);
            break;

        /*
        case PSP_Optical_Flow:
            debug_print("PSP_Optical_Flow (%d)\n", pattern_parm->pattern);
            hipMalloc(&pattern_parm_gpu, sizeof(PixelSortPatternParmOpFlow));
            hipMemcpy(&pattern_parm_gpu, pattern_parm, sizeof(PixelSortPatternParmOpFlow), hipMemcpyHostToDevice);
            break;
         */

        default:
            break;
	}

}



///////////////////////CODE TO DETECT CUDA/////////////////////////////////
/*
for CUDA supporting...

After Effects does not support CUDA based code, if the following functions are there in CUDA programe.
Please remember dont use the below functions while writing the CUDA code.
1) printf()
2) puts()
3) getchar()
4) exit()
5) All timer related functions like cutCreateTimer()
6) dont use CUDA_SAFE_CALL and CUT_SAFE_CALL macros ( which are mostly used in CUDA SDK samples ).
7) CUT_DEVICE_INIT and CUT_EXIT macros
*/

extern "C"
int callCudaFunc();

__global__
void Kernal(int a, int b, int* sum)
{
	*sum = a + b;
}

extern "C"
int callCudaFunc()
{
	// initialise Device
	int deviceCount;
	hipGetDeviceCount(&deviceCount);

	if (deviceCount < 0)
		return 0;

	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, deviceCount);

	// Allocate memory on Device
	int* D_sum = NULL;
	hipError_t error = hipMalloc((void**)&D_sum, sizeof(int) * 1);


	// global function
	Kernal << <1, 1 >> >(1, 2, D_sum);


	// copy data from Device memory to host memory
	int H_sum = 0;
	error = hipMemcpy(&H_sum, D_sum, sizeof(int) * 1, hipMemcpyDeviceToHost);

	// return 
	return H_sum;
}

void cuLayerMemMove(Pixel* GPUinputMem_CPU, Pixel* &GPUinputMem_GPUIn, Pixel* &GPUinputMem_GPUOut, int size,int dir) {
	if (dir == 0) {
		hipMalloc((void**)&GPUinputMem_GPUIn, sizeof(Pixel)*size);
		hipMalloc((void**)&GPUinputMem_GPUOut, sizeof(Pixel)*size);
		hipMemcpy(GPUinputMem_GPUIn, GPUinputMem_CPU, sizeof(Pixel)*size, hipMemcpyHostToDevice);
	}
	else if (dir == 1) {
		hipMemcpy(GPUinputMem_CPU, GPUinputMem_GPUOut, sizeof(Pixel)*size, hipMemcpyDeviceToHost);
		hipFree(GPUinputMem_GPUIn);
		hipFree(GPUinputMem_GPUOut);
	}
}
